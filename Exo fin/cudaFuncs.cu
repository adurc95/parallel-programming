#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "cudaFuncs.h"
#include "constants.h"

__global__ void hist_kernel(const int *data, int *histogram, int size) {
    extern __shared__ int sharedHistogram[];

    // Initialize shared memory
    for (int i = threadIdx.x; i < HISTOGRAM_SIZE; i += blockDim.x) {
        sharedHistogram[i] = 0;
    }
    __syncthreads();

    // Compute histogram in shared memory
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        atomicAdd(&(sharedHistogram[data[tid]]), 1);
    }
    __syncthreads();

    // Accumulate histogram in global memory
    for (int i = threadIdx.x; i < HISTOGRAM_SIZE; i += blockDim.x) {
        atomicAdd(&(histogram[i]), sharedHistogram[i]);
    }
}

int hist_with_cuda(const int *data, int *histogram, int size, int threadsPerBlock, int blocksPerGrid) {
    int *dev_data, *dev_histogram;

    // Allocate GPU memory
    hipMalloc((void **)&dev_data, size * sizeof(int));
    hipMalloc((void **)&dev_histogram, HISTOGRAM_SIZE * sizeof(int));

    // Copy input data to GPU
    hipMemcpy(dev_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    // Initialize histogram on GPU
    hipMemset(dev_histogram, 0, HISTOGRAM_SIZE * sizeof(int));

    // Launch kernel to compute histogram
    hist_kernel<<<blocksPerGrid, threadsPerBlock, HISTOGRAM_SIZE * sizeof(int)>>>(dev_data, dev_histogram, size);

    // Copy histogram back to host
    hipMemcpy(histogram, dev_histogram, HISTOGRAM_SIZE * sizeof(int), hipMemcpyDeviceToHost);

    // Clean up GPU memory
    hipFree(dev_data);
    hipFree(dev_histogram);

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    return 0;
}

